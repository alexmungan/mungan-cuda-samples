#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <float.h>

#include "mm2csr.h"

#include <hipsparse.h>

#include "gpuErrHandler.cuh"
#include "cuBLASErrHandler.cuh"
#include "cuSPARSEErrHandler.cuh"

#include "csr2csc.cuh"
#include "tri_solvers.cuh"

#define eps 1.0e-5
#define loop 1024 //For profiling: number of times to run some operation being profiled in a loop

//Random number generator
double RNG(double min, double max) {
    double scale = rand() / (double)RAND_MAX;  // [0, 1]
    return min + scale * (max - min);          // [min, max]
}

int main() {
	//Host Matrix containers
	double *AA;
	int *IA, *JA, *DA;
	int arrsize, nnz;

	//Get the test matrix 
	char *filepath = "../../data/matrices/sparse/posdef/bmwcra_1.mtx";
	mm2csr(filepath, &AA, &IA, &JA, &DA, &arrsize, &nnz);
	printf("Matrix file read in.\n");
	
	//Device Matrix containers
	double *d_AA;
	int *d_IA, *d_JA, *d_DA;
	gpuErrchk(hipMalloc((void**)&d_AA, nnz*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&d_IA, (arrsize+1)*sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_JA, nnz*sizeof(int)));
	//gpuErrchk(hipMalloc((void**)&d_DA, arrsize*sizeof(int)));	
	//Memcpy to Device
    	gpuErrchk(hipMemcpy(d_AA, AA, nnz*sizeof(double), hipMemcpyHostToDevice));
    	gpuErrchk(hipMemcpy(d_IA, IA, (arrsize+1)*sizeof(int), hipMemcpyHostToDevice));
    	gpuErrchk(hipMemcpy(d_JA, JA, nnz*sizeof(int), hipMemcpyHostToDevice));
    	//gpuErrchk(hipMemcpy(d_DA, DA, arrsize*sizeof(int), hipMemcpyHostToDevice));

	//Generate random RHS for testing
	size_t vecSize = sizeof(double) * arrsize;
	double *r = (double *)malloc(vecSize);
	srand( time(NULL) );
    	for(int i = 0; i < arrsize; i++) r[i] = RNG(-DBL_MAX, DBL_MAX);
    	//for(int i = 0; i < arrsize; i++) printf("r[%d] = %f\n", i, r[i]);

/****	Run CPU solver (results used for correctness checks)	****/
	double *x_correct = (double *)malloc(vecSize);
	
	printf("Running single-threaded CPU solver...");
	clock_t startTime = clock();
	for(int i = 0; i < loop; i++) {
		csr_solve_lower_tri_system(AA, IA, JA, DA, arrsize, x_correct, r);
	}
	clock_t stopTime = clock();
	printf("DONE!\n");
	double cpuTime = ((double)stopTime-startTime)/CLOCKS_PER_SEC;
	cpuTime = cpuTime / loop;
/*******************************************************************/

/****	Profile cuSPARSE tri solver ****/
	double *d_r_cusparse;
    	gpuErrchk(hipMalloc((void**)&d_r_cusparse, vecSize));
    	double *x_cusparse = (double *)malloc(vecSize);
    	double *d_x_cusparse;
    	gpuErrchk(hipMalloc((void**)&d_x_cusparse, vecSize));
    	//cusparse handle
	hipsparseHandle_t handle;
    	cusparseErrchk(hipsparseCreate(&handle));
    	
    	printf("Running cusparse tri solver...");
    	startTime = clock();
    	
    	//cusparse matrix description
    	hipsparseSpMatDescr_t matA;
    	hipsparseDnVecDescr_t vecR;
    	hipsparseDnVecDescr_t vecX;
    	//Create cusparse CSR data structure
    	cusparseErrchk( hipsparseCreateCsr(&matA, arrsize, arrsize, nnz,
                                      d_IA, d_JA, d_AA,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) );
        // Create dense vector r 
        cusparseErrchk( hipsparseCreateDnVec(&vecR, arrsize, d_r_cusparse, HIP_R_64F) );
        // Create dense vector x 
        cusparseErrchk( hipsparseCreateDnVec(&vecX, arrsize, d_x_cusparse, HIP_R_64F) );
        // Create data structure that holds analysis data 
        hipsparseSpSVDescr_t  spsvDescr;
        cusparseErrchk( hipsparseSpSV_createDescr(&spsvDescr) );
        //Set fill mode attribute of matrix description
    	hipsparseFillMode_t fillmode = HIPSPARSE_FILL_MODE_LOWER;
    	cusparseErrchk( hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_FILL_MODE,
                                              &fillmode, sizeof(fillmode)) );
        //Set Unit|Non-Unit diagonal attribute
    	hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_UNIT;
    	cusparseErrchk( hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diagtype, sizeof(diagtype)) );
        //Allocate External buffer for analysis
        void *dBuffer = NULL;
       	 size_t bufferSize = 0;
        double alpha = 1.0;
    	cusparseErrchk( hipsparseSpSV_bufferSize(
                                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, vecR, vecX, HIP_R_64F,
                                HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr,
                                &bufferSize) );
        gpuErrchk( hipMalloc(&dBuffer, bufferSize) );
        
        gpuErrchk(hipMemcpy(d_r_cusparse, r, vecSize, hipMemcpyHostToDevice));
    	gpuErrchk(hipMemset(d_x_cusparse, 0.0f, vecSize));    	
    	cusparseErrchk( hipsparseSpSV_analysis(
                                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, vecR, vecX, HIP_R_64F,
                                HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, dBuffer) );	
	for(int i = 0; i < loop; i++) {
        	// execute SpSV
    		cusparseErrchk( hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha, matA, vecR, vecX, HIP_R_64F,
                                       HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr) );	
	}
	//Memcpy to Host (results)	
	gpuErrchk(hipMemcpy(x_cusparse, d_x_cusparse, vecSize, hipMemcpyDeviceToHost));	
	gpuErrchk(hipFree(dBuffer));
	stopTime = clock();
	printf("DONE!\n");
	double cusparseTriSolverTime = ((double)stopTime-startTime)/CLOCKS_PER_SEC;
	cusparseTriSolverTime = cusparseTriSolverTime / loop;
	
	//free device result
	gpuErrchk(hipFree(d_r_cusparse));
	gpuErrchk(hipFree(d_x_cusparse));
	
	//TEST FOR CORRECTNESS
	bool passed = true;
	for(int i = 0; i < arrsize; i++) {
		if(abs(x_cusparse[i] - x_correct[i]) > eps) {
			passed = false;
			fprintf(stderr, "cuSPARSE_tri_solver failed at cusparse_x[%d] = %f, x[%d] = %f!\n", i, x_cusparse[i], i, x_correct[i]);
		}
	}
	if(passed)
		printf("cuSPARSE_tri_solver PASS\n"); 
/***************************************/

/**** Profile and test CPU vs GPU csr2csc routines ****/
	printf("Testing CPU vs GPU csr2csc routines.\n");
	//CPU ----------------------
	double totalTime = 0;
	
	double *cpu_csc_AA;
    	int *cpu_csc_JA, *cpu_csc_IA, *cpu_colHeadPtrs;
    	
    	for(int i = 0; i < loop; i++) {
    	    	startTime = clock();
    	    	cpu_csc_AA = (double *)malloc(nnz * sizeof(double));
    		cpu_csc_JA = (int *)calloc((arrsize+1), sizeof(int));  //zeroed    		
    		cpu_csc_IA = (int *)malloc(nnz * sizeof(int));	
		cpu_colHeadPtrs = (int *)calloc(arrsize, sizeof(int)); //zeroed
		
    	    	csr2csc(AA, IA, JA, cpu_csc_AA, cpu_csc_JA, cpu_csc_IA, arrsize, nnz, cpu_colHeadPtrs);
    	    	
    	    	stopTime = clock();
    	    	totalTime += (stopTime-startTime);
    	    	
    	    	free(cpu_colHeadPtrs);
    	    	
    	    	//If its the last iteration, keep results so we can use to compare against gpu results
    	    	if(i == (loop-1))
    	    		break;
    	    		
    	    	free(cpu_csc_AA);
		free(cpu_csc_JA);
		free(cpu_csc_IA);
    	}
    	

    	double cpu_csr2csc_time = ((double)totalTime)/CLOCKS_PER_SEC;
    	cpu_csr2csc_time = cpu_csr2csc_time / loop;
    	
    	//GPU ---------------------- NOTE: this gpu routine could possibly be significantly improved through the use of CUDA streams to hide memcpy
    	totalTime = 0;
    	//Gpu CSR
    	double *gpu_AA; 
    	int *gpu_IA, *gpu_JA;   
    	//GPU CSC	
    	double *gpu_csc_AA;
    	int *gpu_csc_JA, *gpu_csc_IA, *gpu_colHeadPtrs;
    	
    	for(int i = 0; i < loop; i++) {
    	    	startTime = clock();
    	    	//GPU CSR 
    	    	gpuErrchk(hipMalloc((void**)&gpu_AA, nnz*sizeof(double)));
    	    	gpuErrchk(hipMalloc((void**)&gpu_IA, (arrsize+1)*sizeof(int)));
    	    	gpuErrchk(hipMalloc((void**)&gpu_JA, nnz*sizeof(int)));   	 	
    		gpuErrchk(hipMemcpy(gpu_AA, AA, nnz*sizeof(double), hipMemcpyHostToDevice));
    		gpuErrchk(hipMemcpy(gpu_IA, IA, (arrsize+1)*sizeof(int), hipMemcpyHostToDevice));
    		gpuErrchk(hipMemcpy(gpu_JA, JA, nnz*sizeof(int), hipMemcpyHostToDevice));
     	
    	    	//GPU CSC containers
    	    	gpuErrchk(hipMalloc((void**)&gpu_csc_AA, nnz*sizeof(double)));
    	    	gpuErrchk(hipMalloc((void**)&gpu_csc_JA, (arrsize+1)*sizeof(int)));
    	    	gpuErrchk(hipMemset(gpu_csc_JA, 0, (arrsize+1)*sizeof(int)));  //zeroed
    	    	gpuErrchk(hipMalloc((void**)&gpu_csc_IA, nnz*sizeof(int)));
    	    	gpuErrchk(hipMalloc((void**)&gpu_colHeadPtrs, arrsize * sizeof(int)));
    	    	gpuErrchk(hipMemset(gpu_colHeadPtrs, 0, arrsize*sizeof(int))); //zeroed
    	    	
    	    	/* KERNEL */
		int blockSize = 128;
		dim3 nnzGridSize(((nnz + blockSize - 1)/ blockSize),1,1);
		cu_csr2csc_part1<<<nnzGridSize, blockSize>>>(gpu_JA, gpu_csc_JA, nnz);
		
		int *host_csc_JA = (int *)malloc((arrsize+1)*sizeof(int));
		gpuErrchk(hipMemcpy(host_csc_JA, gpu_csc_JA, (arrsize+1)*sizeof(int), hipMemcpyDeviceToHost));
		cu_csr2csc_part2(host_csc_JA, arrsize);
		gpuErrchk(hipMemcpy(gpu_csc_JA, host_csc_JA, (arrsize+1)*sizeof(int), hipMemcpyHostToDevice));
		free(host_csc_JA);
		
		dim3 arrsizeGridSize(((arrsize + blockSize - 1)/ blockSize),1,1);
		for(int rowidx = 0; rowidx < arrsize; rowidx++) {
			cu_csr2csc_part3<<<arrsizeGridSize, blockSize>>>(gpu_AA, gpu_IA, gpu_JA, gpu_csc_AA, gpu_csc_JA, gpu_csc_IA, gpu_colHeadPtrs, rowidx);
			//implicit global sync for each kernel call
			//Basic Kernel Error Checking
			gpuErrchk( hipPeekAtLastError() );
        		gpuErrchk( hipDeviceSynchronize() );
		}	
    	    	/**********/   	    	
    	    	
    	    	stopTime = clock();
    	    	totalTime += (stopTime-startTime);
    	    	
    	    	gpuErrchk(hipFree(gpu_AA));
    	    	gpuErrchk(hipFree(gpu_IA));
    	    	gpuErrchk(hipFree(gpu_JA));
    	    	gpuErrchk(hipFree(gpu_colHeadPtrs));
    	    	
    	    	//If its the last iteration, don't free csc mem, so we can check gpu results
    	    	if(i == (loop-1))
    	    		break;
    	    	
    	    	gpuErrchk(hipFree(gpu_csc_AA));
    	    	gpuErrchk(hipFree(gpu_csc_JA));
    	    	gpuErrchk(hipFree(gpu_csc_IA));
    	    			
    	}
    	
    	//GPU Correctness Check -----
    	double *test_csc_AA = (double *)malloc(nnz * sizeof(double));
    	int *test_csc_JA = (int *)malloc((arrsize+1) * sizeof(int));    		
    	int *test_csc_IA = (int *)malloc(nnz * sizeof(int));	
    	gpuErrchk(hipMemcpy(test_csc_AA, gpu_csc_AA, nnz*sizeof(double), hipMemcpyDeviceToHost));
    	gpuErrchk(hipMemcpy(test_csc_JA, gpu_csc_JA, (arrsize+1) * sizeof(int), hipMemcpyDeviceToHost));
    	gpuErrchk(hipMemcpy(test_csc_IA, gpu_csc_IA, nnz * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(gpu_csc_AA));
    	gpuErrchk(hipFree(gpu_csc_JA));
    	gpuErrchk(hipFree(gpu_csc_IA));
    	
    	//check
    	passed = true;
    	for(int i = 0; i < nnz; i++) {
    		if(abs(test_csc_AA[i] - cpu_csc_AA[i]) > eps) {
    			fprintf(stderr, "csr2csc test failed! CSC values arrays do not match --> \
    			CPU_AA[%d] = %f. GPU_AA[%d] = %f\n", i, cpu_csc_AA[i], i, test_csc_AA[i]);
    			passed = false;
    		}
    		if(abs(test_csc_IA[i] - cpu_csc_IA[i]) > eps) {
    			fprintf(stderr, "csr2csc test failed! CSC rowIndicies arrays do not match --> \
    			CPU_IA[%d] = %d. GPU_IA[%d] = %d\n", i, cpu_csc_IA[i], i, test_csc_IA[i]);
    			    			passed = false;
    		}
    	}
    	for(int i = 0; i < arrsize+1; i++) {
    		if(abs(test_csc_JA[i] - cpu_csc_JA[i]) > eps) {
  	    		fprintf(stderr, "csr2csc test failed! CSC colPtrs arrays do not match --> \
    			CPU_AA[%d] = %d. GPU_AA[%d] = %d\n", i, cpu_csc_JA[i], i, test_csc_JA[i]);
    			    			passed = false;
    		}
    	}
    	if(!passed)
    		printf("gpu csr2csc failed!\n");
    	else
    		printf("gpu csc2csc PASS!\n");

    	
    	free(cpu_csc_AA);
	free(cpu_csc_JA);
	free(cpu_csc_IA);
	free(test_csc_AA);
	free(test_csc_JA);
	free(test_csc_IA);
	//----------------------------

    	double gpu_csr2csc_time = ((double)totalTime)/CLOCKS_PER_SEC;
    	gpu_csr2csc_time = gpu_csr2csc_time / loop;
    	printf("CPU csr2csc execution time: %fs\n", cpu_csr2csc_time);
    	printf("GPU csr2csc execution time: %fs\n", gpu_csr2csc_time);

/******************************************************/

/****	Profile my GPU only tri solver ****/
	printf("Running myGpuSolver_1...");
	double *x_mygpu = (double *)malloc(vecSize);
	//TODO: 0 initialize?????
	
    	startTime = clock();
    	
    	//Memcopies 
    	
    	//Convert to CSC for fast col access
    	//double *csc_AA = (double *)malloc(nnz * sizeof(double));
    	//int *csc_JA = (int *)malloc((arrsize+1) * sizeof(int));
    	//int *csc_IA = (int *)malloc(nnz * sizeof(int));	
    	//csr2csc(AA, IA, JA, csc_AA, csc_JA, csc_IA, arrsize, nnz);
    	//printf("Matrix converted to CSC format!\n");
    	
    	//
    	
	
	stopTime = clock();
	printf("DONE!\n");
	double myGpuSolverTime = ((double)stopTime-startTime)/CLOCKS_PER_SEC;
	myGpuSolverTime = myGpuSolverTime / loop;

/******************************************/

	
/****	Output Results for Unit Triangular Solvers	****/
	printf("(Single-threaded) CPU_tri_solver execution time: %fs\n", cpuTime);
	printf("cuSPARSE_tri_solver execution time: %fs\n", cusparseTriSolverTime);
	printf("myGpuSolver_1 execution time: %fs\n", myGpuSolverTime);
/***********************************************************/
	
/****	Free Resources	****/
	//Free host matrix
	free(AA);
	free(IA);
	free(JA);
	free(DA);
	//Free device matrix
	gpuErrchk(hipFree(d_AA));
	gpuErrchk(hipFree(d_IA));
	gpuErrchk(hipFree(d_JA));
	//gpuErrchk(hipFree(d_DA));
	//Free RHS
	free(r);
	free(x_correct);
	free(x_cusparse);
	//destroy matrix/vector descriptors
    	cusparseErrchk( hipsparseDestroySpMat(matA) );
    	cusparseErrchk( hipsparseDestroyDnVec(vecR) );
    	cusparseErrchk( hipsparseDestroyDnVec(vecX) );
    	cusparseErrchk( hipsparseSpSV_destroyDescr(spsvDescr));
    	cusparseErrchk(hipsparseDestroy(handle));
/***************************/


	return 0;
}
