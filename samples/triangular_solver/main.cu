#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <float.h>

#include "mm2csr.h"

#include <hipsparse.h>

#include "gpuErrHandler.cuh"
#include "cuBLASErrHandler.cuh"
#include "cuSPARSEErrHandler.cuh"

#define eps 1.0e-5
#define loop 1024

//CPU SOLVER: used for correctness checks
void csr_solve_lower_tri_system(double AA[], int IA[], int JA[], int DA[], int nn, double x[], double r[])
{
// Purpose: compute (I + Low_tri(A))*x, return modified x.
// x[] input as the rhs vector, output as the solution.

    int idx,k1,k2,k,j;

    for (idx = 0; idx < nn; idx++) {   // compressed sparse row format
        k1 = IA[idx];
        k2 = DA[idx] - 1;
        x[idx] = r[idx];
        for (k = k1; k <= k2; k++) {
            j = JA[k];
            x[idx] -= AA[k]*x[j];
        }
    }
}

//Random number generator
double RNG(double min, double max) {
    double scale = rand() / (double)RAND_MAX;  // [0, 1]
    return min + scale * (max - min);          // [min, max]
}

int main() {
	//Host Matrix containers
	double *AA;
	int *IA, *JA, *DA;
	int arrsize, nnz;

	//Get the test matrix 
	char *filepath = "../../data/matrices/sparse/posdef/bmwcra_1.mtx";
	mm2csr(filepath, &AA, &IA, &JA, &DA, &arrsize, &nnz);
	printf("Matrix file read in.\n");
	
	//Device Matrix containers
	double *d_AA;
	int *d_IA, *d_JA, *d_DA;
	gpuErrchk(hipMalloc((void**)&d_AA, nnz*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&d_IA, (arrsize+1)*sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_JA, nnz*sizeof(int)));
	//gpuErrchk(hipMalloc((void**)&d_DA, arrsize*sizeof(int)));	
	//Memcpy to Device
    	gpuErrchk(hipMemcpy(d_AA, AA, nnz*sizeof(double), hipMemcpyHostToDevice));
    	gpuErrchk(hipMemcpy(d_IA, IA, (arrsize+1)*sizeof(int), hipMemcpyHostToDevice));
    	gpuErrchk(hipMemcpy(d_JA, JA, nnz*sizeof(int), hipMemcpyHostToDevice));
    	//gpuErrchk(hipMemcpy(d_DA, DA, arrsize*sizeof(int), hipMemcpyHostToDevice));

	//Generate random RHS for testing
	size_t vecSize = sizeof(double) * arrsize;
	double *r = (double *)malloc(vecSize);
	srand( time(NULL) );
    	for(int i = 0; i < arrsize; i++) r[i] = RNG(-DBL_MAX, DBL_MAX);
    	//for(int i = 0; i < arrsize; i++) printf("r[%d] = %f\n", i, r[i]);

/****	Run CPU solver (results used for correctness checks)	****/
	double *x_correct = (double *)malloc(vecSize);
	
	clock_t startTime = clock();
	for(int i = 0; i < loop; i++) {
		csr_solve_lower_tri_system(AA, IA, JA, DA, arrsize, x_correct, r);
	}
	clock_t stopTime = clock();
	double cpuTime = ((double)stopTime-startTime)/CLOCKS_PER_SEC;
	cpuTime = cpuTime / loop;
/*******************************************************************/

/****	Profile cuSPARSE tri solver ****/
	double *d_r_cusparse;
    	gpuErrchk(hipMalloc((void**)&d_r_cusparse, vecSize));
    	double *x_cusparse = (double *)malloc(vecSize);
    	double *d_x_cusparse;
    	gpuErrchk(hipMalloc((void**)&d_x_cusparse, vecSize));
    	//cusparse handle
	hipsparseHandle_t handle;
    	cusparseErrchk(hipsparseCreate(&handle));
    	
    	startTime = clock();
    	
    	//cusparse matrix description
    	hipsparseSpMatDescr_t matA;
    	hipsparseDnVecDescr_t vecR;
    	hipsparseDnVecDescr_t vecX;
    	//Create cusparse CSR data structure
    	cusparseErrchk( hipsparseCreateCsr(&matA, arrsize, arrsize, nnz,
                                      d_IA, d_JA, d_AA,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) );
        // Create dense vector r 
        cusparseErrchk( hipsparseCreateDnVec(&vecR, arrsize, d_r_cusparse, HIP_R_64F) );
        // Create dense vector x 
        cusparseErrchk( hipsparseCreateDnVec(&vecX, arrsize, d_x_cusparse, HIP_R_64F) );
        // Create data structure that holds analysis data 
        hipsparseSpSVDescr_t  spsvDescr;
        cusparseErrchk( hipsparseSpSV_createDescr(&spsvDescr) );
        //Set fill mode attribute of matrix description
    	hipsparseFillMode_t fillmode = HIPSPARSE_FILL_MODE_LOWER;
    	cusparseErrchk( hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_FILL_MODE,
                                              &fillmode, sizeof(fillmode)) );
        //Set Unit|Non-Unit diagonal attribute
    	hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_UNIT;
    	cusparseErrchk( hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diagtype, sizeof(diagtype)) );
        //Allocate External buffer for analysis
        void *dBuffer = NULL;
       	 size_t bufferSize = 0;
        double alpha = 1.0;
    	cusparseErrchk( hipsparseSpSV_bufferSize(
                                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, vecR, vecX, HIP_R_64F,
                                HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr,
                                &bufferSize) );
        gpuErrchk( hipMalloc(&dBuffer, bufferSize) );
        
        gpuErrchk(hipMemcpy(d_r_cusparse, r, vecSize, hipMemcpyHostToDevice));
    	gpuErrchk(hipMemset(d_x_cusparse, 0.0f, vecSize));    	
    	cusparseErrchk( hipsparseSpSV_analysis(
                                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, vecR, vecX, HIP_R_64F,
                                HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, dBuffer) );	
	for(int i = 0; i < loop; i++) {
        	// execute SpSV
    		cusparseErrchk( hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha, matA, vecR, vecX, HIP_R_64F,
                                       HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr) );	
	}
	//Memcpy to Host (results)	
	gpuErrchk(hipMemcpy(x_cusparse, d_x_cusparse, vecSize, hipMemcpyDeviceToHost));	
	gpuErrchk(hipFree(dBuffer));
	stopTime = clock();
	double cusparseTriSolverTime = ((double)stopTime-startTime)/CLOCKS_PER_SEC;
	cusparseTriSolverTime = cusparseTriSolverTime / loop;
	
	//free device result
	gpuErrchk(hipFree(d_r_cusparse));
	gpuErrchk(hipFree(d_x_cusparse));
	
	//TEST FOR CORRECTNESS
	bool passed = true;
	for(int i = 0; i < arrsize; i++) {
		if(abs(x_cusparse[i] - x_correct[i]) > eps) {
			passed = false;
			fprintf(stderr, "cuSPARSE_tri_solver failed at cusparse_x[%d] = %f, x[%d] = %f!\n", i, x_cusparse[i], i, x_correct[i]);
		}
	}
	if(passed)
		printf("cuSPARSE_tri_solver PASS\n"); 
/***************************************/
	
/****	Output Results for Unit Triangular Solvers	****/
	printf("(Single-threaded) CPU_tri_solver execution time: %fms\n", cpuTime);
	printf("cuSPARSE_tri_solver execution time: %fms\n", cusparseTriSolverTime);
/***********************************************************/
	
/****	Free Resources	****/
	//Free host matrix
	free(AA);
	free(IA);
	free(JA);
	free(DA);
	//Free device matrix
	gpuErrchk(hipFree(d_AA));
	gpuErrchk(hipFree(d_IA));
	gpuErrchk(hipFree(d_JA));
	//gpuErrchk(hipFree(d_DA));
	//Free RHS
	free(r);
	free(x_correct);
	free(x_cusparse);
	//destroy matrix/vector descriptors
    	cusparseErrchk( hipsparseDestroySpMat(matA) );
    	cusparseErrchk( hipsparseDestroyDnVec(vecR) );
    	cusparseErrchk( hipsparseDestroyDnVec(vecX) );
    	cusparseErrchk( hipsparseSpSV_destroyDescr(spsvDescr));
    	cusparseErrchk(hipsparseDestroy(handle));
/***************************/


	return 0;
}
