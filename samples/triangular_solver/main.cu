#include "hip/hip_runtime.h"
/* Contains main() which calls the tri solve kernels */
/* Their are multiple implementations of tri solve - this file tests and profiles them */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "mm2csr.h"

#include <hipblas.h>
#include <hipsparse.h>

#include "gpuErrHandler.cuh"
#include "cuBLASErrHandler.cuh"
#include "cuSPARSEErrHandler.cuh"

/* Warm up kernel - wake gpu up before profiling */
__global__ void warm_up_gpu(){
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid; 
}

int main() {
	//Host Matrix containers
	double *AA;
	int *IA, *JA, *DA;
	int arrsize, nnz;

	//Get the test matrix 
	char *filepath = "../../data/matrices/sparse/posdef/bcsstk13.mtx";
	mm2csr(filepath, &AA, &IA, &JA, &DA, &arrsize, &nnz);
	printf("Matrix file read in.\n");
	
	//Device Matrix containers
	double *d_AA;
	int *d_IA, *d_JA, *d_DA;
	gpuErrchk(hipMalloc((void**)&d_AA, nnz*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&d_IA, (arrsize+1)*sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_JA, nnz*sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_DA, arrsize*sizeof(int)));	
	//Memcpy to Device
    	gpuErrchk(hipMemcpy(d_AA, AA, nnz*sizeof(double), hipMemcpyHostToDevice));
    	gpuErrchk(hipMemcpy(d_IA, IA, (arrsize+1)*sizeof(int), hipMemcpyHostToDevice));
    	gpuErrchk(hipMemcpy(d_JA, JA, nnz*sizeof(int), hipMemcpyHostToDevice));
    	gpuErrchk(hipMemcpy(d_DA, DA, arrsize*sizeof(int), hipMemcpyHostToDevice));

	//Generate random RHS for testing
	size_t vecSize = sizeof(double) * arrsize;
	double *r = (double *)malloc(vecSize);
	srand( time(NULL) );
    	for(int i = 0; i < arrsize; i++) r[i] = rand() / (double)RAND_MAX;

	//Warm up kernel
	dim3 numThreads(128,1,1);
	dim3 numBlocks(128,1,1);
	warm_up_gpu<<<numBlocks, numThreads>>>();

/****	Profile cuSPARSE tri solver ****/
	double *r_cusparse = (double *)malloc(vecSize);
	memcpy(r_cusparse, r, vecSize);
	double *d_r_cusparse;
    	gpuErrchk(hipMalloc((void**)&d_r_cusparse, vecSize));
    	clock_t startTime = clock();
    	gpuErrchk(hipMemcpy(d_r_cusparse, r_cusparse, vecSize, hipMemcpyHostToDevice));
	//cusparse handle
	hipsparseHandle_t handle;
    	cusparseErrchk(hipsparseCreate(&handle));
    	//cusparse matrix description
    	hipsparseSpMatDescr_t matA;
    	hipsparseDnVecDescr_t vecInOut;
    	//Create cusparse CSR data structure
    	cusparseErrchk( hipsparseCreateCsr(&matA, arrsize, arrsize, nnz,
                                      d_IA, d_JA, d_AA,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) );
        // Create dense vector r 
        cusparseErrchk( hipsparseCreateDnVec(&vecInOut, arrsize, d_r_cusparse, HIP_R_64F) );
        // Create data structure that holds analysis data 
        hipsparseSpSVDescr_t  spsvDescr;
        cusparseErrchk( hipsparseSpSV_createDescr(&spsvDescr) );
        //Set fill mode attribute of matrix description
    	hipsparseFillMode_t fillmode = HIPSPARSE_FILL_MODE_LOWER;
    	cusparseErrchk( hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_FILL_MODE,
                                              &fillmode, sizeof(fillmode)) );
        //Set Unit|Non-Unit diagonal attribute
    	hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
    	cusparseErrchk( hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diagtype, sizeof(diagtype)) );
        //Allocate External buffer for analysis
        void *dBuffer = NULL;
        size_t bufferSize = 0;
        float alpha = 1.0;
    	cusparseErrchk( hipsparseSpSV_bufferSize(
                                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, vecInOut, vecInOut, HIP_R_64F,
                                HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr,
                                &bufferSize) );
        gpuErrchk( hipMalloc(&dBuffer, bufferSize) );
	cusparseErrchk( hipsparseSpSV_analysis(
                                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, vecInOut, vecInOut, HIP_R_64F,
                                HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, dBuffer) );
        // execute SpSV
    	cusparseErrchk( hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha, matA, vecInOut, vecInOut, HIP_R_64F,
                                       HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr) );
        //Memcpy to Host (results)	
	gpuErrchk(hipMemcpy(r_cusparse, d_r_cusparse, vecSize, hipMemcpyDeviceToHost));
	
	clock_t stopTime = clock();
	double cusparseTriSolverTime = ((double)stopTime-startTime)/CLOCKS_PER_SEC;
	//free device result
	gpuErrchk(hipFree(d_r_cusparse));
	
	//TEST FOR CORRECTNESS????????????????????????????//
/***************************************/
	
/****	Output Results for Unit Triangular Solvers	****/
	printf("cuSPARSE_tri_solver execution time: %fms\n", cusparseTriSolverTime);
/***********************************************************/
	
/****	Free Resources	****/
	//Free host matrix
	free(AA);
	free(IA);
	free(JA);
	free(DA);
	//Free device matrix
	gpuErrchk(hipFree(d_AA));
	gpuErrchk(hipFree(d_IA));
	gpuErrchk(hipFree(d_JA));
	gpuErrchk(hipFree(d_DA));
	//Free RHS
	free(r);
	free(r_cusparse);
	 // destroy matrix/vector descriptors
    	cusparseErrchk( hipsparseDestroySpMat(matA) );
    	cusparseErrchk( hipsparseDestroyDnVec(vecInOut) );
    	cusparseErrchk( hipsparseSpSV_destroyDescr(spsvDescr));
    	cusparseErrchk(hipsparseDestroy(handle));
/***************************/


	return 0;
}
